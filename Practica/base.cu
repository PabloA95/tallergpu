#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <sys/resource.h>

#ifdef _INT_
typedef int basetype;     // Tipo para elementos: int
#define labelelem    "ints"
#elif _DOUBLE_
typedef double basetype;  // Tipo para elementos: double
#define labelelem    "doubles"
#else
typedef float basetype;   // Tipo para elementos: float     PREDETERMINADO
#define labelelem    "floats"
#endif

double dwalltime(){
        double sec;
        struct timeval tv;
        gettimeofday(&tv,NULL);
        sec = tv.tv_sec + tv.tv_usec/1000000.0;
        return sec;
}

/*
Función para inicializar el vector que vamos a utilizar
*/
void init_CPU_array(basetype array[], const unsigned int n)
{
        // unsigned int i;
        // for(i = 0; i < n; i++) {
        //         array[i] = (basetype)i;
        // }
}

void funcion_CPU(basetype vec[], const unsigned int c, const unsigned int n) {
        // Codigo
}

__global__ void funcion_kernel_cuda(basetype *const arrayV,   const int n, const int c){
        unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_id < n)
        arrayV[global_id] = arrayV[global_id]*c;
}

void funcion_GPU( basetype arrayV[], const unsigned int n, const unsigned int blk_size, const int c){
        double timetick;
        // Número de bytes de cada uno de nuestros vectores
        unsigned int numBytes = n * sizeof(basetype);
        hipError_t error;

        // Reservamos memoria global del device (GPU) para el array y lo copiamos
        basetype *cV;
        timetick = dwalltime();
        hipMalloc((void **) &cV, numBytes);
        printf("-> Tiempo de alocacion en memoria global de GPU %f\n", dwalltime() - timetick);  
        timetick = dwalltime();
        hipMemcpy(cV, arrayV, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
        printf("-> Tiempo de copia de memoria CPU =>> GPU %f\n", dwalltime() - timetick);

        // Bloque unidimensional de hilos (*blk_size* hilos)
        dim3 dimBlock(blk_size);
        // Grid unidimensional (*ceil(n/blk_size)* bloques)
        dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x);
        
        // Lanzamos ejecución del kernel en la GPU
        // timestamp(start);            // Medimos tiempo de cálculo en GPU
        timetick = dwalltime();
        funcion_kernel_cuda<<<dimGrid, dimBlock>>>(cV, n, c);
        error = hipDeviceSynchronize();
        printf("Synchronyse error: %d\n", error);
        printf("-> Tiempo de ejecucion en GPU %f\n", dwalltime() - timetick);
        //timestamp(end);

        // Movemos resultado: GPU -> CPU
        timetick = dwalltime();
        hipMemcpy(arrayV, cV, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
        printf("-> Tiempo de copia GPU ==>> CPU %f\n", dwalltime() - timetick);

        // Liberamos memoria global del device utilizada
        hipFree (cV);
}



int main(int argc, char **argv)
{
        double timetick;
        // Ejecucion en CPU
        basetype *vec;
        unsigned int blk_size;
        int c;

        timetick = dwalltime();
        // Alocar e inicializar vector y variables
        printf("-> Tiempo de inicializacion de vector en CPU %f\n", dwalltime() - timetick);

        timetick = dwalltime();
        // Funcion que se ejecuta en la CPU
        printf("-> Tiempo de ejecucion en CPU %f\n", dwalltime() - timetick);

        // Ejecucion en GPU
        // Inicializa nuevamente el vector para realizar la ejecucion en GPU
        // Funcion que se ejecuta en la GPU
        // Chequea si el resultado obtenido en la GPU es correcto
        // check_array(vec,n);

        free(vec);
        return 0;
}
